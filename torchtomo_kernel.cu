#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

inline int roundup_div(const int x, const int y) {
    return x / y + (x % y != 0);
}

// computes one projetion view
__global__ void projection_view_kernel(
                    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> image,
                    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const int nbins,
                    const int nviews,
                    const int nx,
                    const int ny){

  const int uindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int sindex = blockIdx.y * blockDim.y + threadIdx.y;
  const int ib = blockIdx.z;

  //compute projection for a single ray
  if ((uindex < nbins) && (sindex < nviews)) {
    const float s = sindex*ds;

    // location of the source
    const float xsource = radius*cos(s);
    const float ysource = radius*sin(s);

    // detector center
    const float xDetCenter = (radius - source_to_detector)*cos(s);
    const float yDetCenter = (radius - source_to_detector)*sin(s);

    // unit vector in the direction of the detector line
    const float eux = -sin(s);
    const float euy =  cos(s);

    const float u = u0 + (uindex+0.5)*du;
    const float xbin = xDetCenter + eux*u;
    const float ybin = yDetCenter + euy*u;

    const float xl = x0;
    const float yl = y0;

    const float xdiff = xbin-xsource;
    const float ydiff = ybin-ysource;
    const float xad = abs(xdiff)*dy;
    const float yad = abs(ydiff)*dx;

    float raysum = 0.0f; // acculumator variable

    if (xad > yad){  // loop through x-layers of image if xad>yad. This ensures ray hits only one or two pixels per layer
      float slope = ydiff/xdiff;
      float travPixlen = dx*sqrt(1.0f+slope*slope);
      float yIntOld = ysource+slope*(xl-xsource);
      int iyOld = static_cast<int>(floor((yIntOld-y0)/dy));
      // loop over x-layers
      for (int ix = 0; ix < nx; ix++){
         float x=xl+dx*(ix + 1.0f);
         float yIntercept=ysource+slope*(x-xsource);
         int iy = static_cast<int>(floor((yIntercept-y0)/dy));
         if (iy == iyOld){ // if true, ray stays in the same pixel for this x-layer
            if ((iy >= 0) && (iy < ny)) {
               raysum += travPixlen*image[ib][ix][iy];
            }
         } else {    // else case is if ray hits two pixels for this x-layer
            float yMid=dy*max(iy,iyOld)+yl;
            float ydist1=abs(yMid-yIntOld);
            float ydist2=abs(yIntercept-yMid);
            float frac1=ydist1/(ydist1+ydist2);
            float frac2=1.0f-frac1;
            if ((iyOld >= 0) && (iyOld < ny)){
               raysum += frac1*travPixlen*image[ib][ix][iyOld];
             }
            if ((iy>=0) && (iy<ny)){
               raysum += frac2*travPixlen*image[ib][ix][iy];
             }
         }
         iyOld=iy;
         yIntOld=yIntercept;
       }

    } else {// through y-layers of image if xad<=yad
      float slopeinv=xdiff/ydiff;
      float travPixlen=dy*sqrt(1.0+slopeinv*slopeinv);
      float xIntOld=xsource+slopeinv*(yl-ysource);
      int ixOld= static_cast<int>(floor((xIntOld-x0)/dx));
      // loop over y-layers
      for (int iy = 0; iy < ny; iy++){
         float y=yl+dy*(iy + 1.0);
         float xIntercept=xsource+slopeinv*(y-ysource);
         int ix = static_cast<int>(floor((xIntercept-x0)/dx));
         if (ix == ixOld){// if true, ray stays in the same pixel for this y-layer
            if ((ix >= 0) && (ix < nx)){
               raysum += travPixlen*image[ib][ix][iy];
             }
         } else {  // else case is if ray hits two pixels for this y-layer
            float xMid=dx*max(ix,ixOld)+xl;
            float xdist1=abs(xMid-xIntOld);
            float xdist2=abs(xIntercept-xMid);
            float frac1=xdist1/(xdist1+xdist2);
            float frac2=1.0-frac1;
            if ((ixOld >= 0) && (ixOld < nx)){
               raysum += frac1*travPixlen*image[ib][ixOld][iy];
            }
            if ((ix>=0) && (ix<nx)){
               raysum += frac2*travPixlen*image[ib][ix][iy];
            }
         }
         ixOld = ix;
         xIntOld = xIntercept;
       }
    }
    sinogram[ib][sindex][uindex]=raysum;
 }
}

// computes one backprojection view
__global__ void backprojection_view_kernel(
                    torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> image,
                    const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> sinogram,
                    const float dx,
                    const float dy,
                    const float x0,
                    const float y0,
                    const float fanangle2,
                    const float detectorlength,
                    const float u0,
                    const float du,
                    const float ds,
                    const float radius,
                    const float source_to_detector,
                    const float fov_radius,
                    const int nbins,
                    const int nviews,
                    const int nx,
                    const int ny){

  const int uindex = blockIdx.x * blockDim.x + threadIdx.x; //detector index
  const int sindex = blockIdx.y * blockDim.y + threadIdx.y; //view index
  const int ib = blockIdx.z; //batch index

  //compute backprojection for a single ray
  if ((uindex < nbins) && (sindex < nviews)) {
    const auto sinoval = sinogram[ib][sindex][uindex];

    const float s = sindex*ds;

    // location of the source
    const float xsource = radius*cos(s);
    const float ysource = radius*sin(s);

    // detector center
    const float xDetCenter = (radius - source_to_detector)*cos(s);
    const float yDetCenter = (radius - source_to_detector)*sin(s);

    // unit vector in the direction of the detector line
    const float eux = -sin(s);
    const float euy =  cos(s);

    const float fov_radius2 = fov_radius*fov_radius; //used to set image mask

    const float u = u0+(uindex+0.5)*du;
    const float xbin = xDetCenter + eux*u;
    const float ybin = yDetCenter + euy*u;

    const float xl=x0;
    const float yl=y0;

    const float xdiff=xbin-xsource;
    const float ydiff=ybin-ysource;
    const float xad=abs(xdiff)*dy;
    const float yad=abs(ydiff)*dx;

    if (xad>yad){   // loop through x-layers of image if xad>yad. This ensures ray hits only one or two pixels per layer
       float slope=ydiff/xdiff;
       float travPixlen=dx*sqrt(1.0+slope*slope);
       float yIntOld=ysource + slope*(xl-xsource);
       int iyOld = static_cast<int>(floor((yIntOld-y0)/dy));
       // loop over x-layers
       for (int ix = 0; ix < nx; ix++){
          float x = xl + dx*(ix + 1.0);
          float yIntercept=ysource+slope*(x-xsource);
          int iy = static_cast<int>(floor((yIntercept-y0)/dy));
          float pix_x = x0 + dx*(ix+0.5); //used to set mask
          float pix_y = y0 + dy*(iy+0.5); //used to set mask
          float pix_y_old = y0 + dy*(iyOld+0.5); // used to set mask
            if (iy == iyOld){ // if true, ray stays in the same pixel for this x-layer
             if ((iy >= 0) && (iy < ny) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)){
                atomicAdd(&image[ib][ix][iy],sinoval*travPixlen);
                // image[ib][ix][iy] += sinoval*travPixlen;
              }
          } else {    // else case is if ray hits two pixels for this x-layer
             float yMid = dy*max(iy,iyOld)+yl;
             float ydist1 = abs(yMid-yIntOld);
             float ydist2 = abs(yIntercept-yMid);
             float frac1 = ydist1/(ydist1+ydist2);
             float frac2 = 1.0-frac1;
             if ((iyOld >= 0) && (iyOld < ny) && (pix_x*pix_x + pix_y_old*pix_y_old <= fov_radius2)){
                atomicAdd(&image[ib][ix][iyOld],frac1*sinoval*travPixlen);
                // image[ib][ix][iyOld] += frac1*sinoval*travPixlen;
              }
             if ((iy >= 0) && (iy < ny) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)) {
                atomicAdd(&image[ib][ix][iy],frac2*sinoval*travPixlen);
                // image[ib][ix][iy] += frac2*sinoval*travPixlen;
              }
          }
          iyOld=iy;
          yIntOld=yIntercept;
        }
    } else { //loop through y-layers of image if xad<=yad
       float slopeinv=xdiff/ydiff;
       float travPixlen=dy*sqrt(1.0+slopeinv*slopeinv);
       float xIntOld=xsource+slopeinv*(yl-ysource);
       int ixOld = static_cast<int>(floor((xIntOld-x0)/dx));
       for (int iy = 0; iy < ny; iy++){
          float y = yl + dy*(iy + 1.0);
          float xIntercept = xsource+slopeinv*(y-ysource);
          int ix = static_cast<int>(floor((xIntercept-x0)/dx));
          float pix_x = x0 + dx*(ix+0.5);
          float pix_y = y0 + dy*(iy+0.5);
          float pix_x_old = x0 + dx*(ixOld+0.5); // used to set mask
          if (ix == ixOld){ // if true, ray stays in the same pixel for this y-layer
             if ((ix >= 0) && (ix < nx) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)) {
                atomicAdd(&image[ib][ix][iy],sinoval*travPixlen);
                // image[ib][ix][iy] += sinoval*travPixlen;
              }
          } else { // else case is if ray hits two pixels for this y-layer
             float xMid = dx*max(ix,ixOld)+xl;
             float xdist1 = abs(xMid-xIntOld);
             float xdist2 = abs(xIntercept-xMid);
             float frac1 = xdist1/(xdist1+xdist2);
             float frac2=1.0-frac1;
             if ((ixOld >= 0) && (ixOld < nx) && (pix_x_old*pix_x_old + pix_y*pix_y <= fov_radius2)){
                atomicAdd(&image[ib][ixOld][iy],frac1*sinoval*travPixlen);
                // image[ib][ixOld][iy] += frac1*sinoval*travPixlen;
              }
             if ((ix >= 0) && (ix < nx) && (pix_x*pix_x + pix_y*pix_y <= fov_radius2)){
                atomicAdd(&image[ib][ix][iy],frac2*sinoval*travPixlen);
                // image[ib][ix][iy] += frac2*sinoval*travPixlen;
              }
          }
          ixOld = ix;
          xIntOld = xIntercept;
       }
     }
   } // end main block
}

// computes pixel-driven projetion
__global__ void projection_wpd_kernel(const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> image,
                                          torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> sinogram,
                                          const float dx,
                                          const float dy,
                                          const float x0,
                                          const float y0,
                                          const float fanangle2,
                                          const float detectorlength,
                                          const float u0,
                                          const float du,
                                          const float ds,
                                          const float radius,
                                          const float source_to_detector,
                                          const float fov_radius,
                                          const int nbins,
                                          const int nviews,
                                          const int nx,
                                          const int ny){

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int ib = blockIdx.z; //batch index

    if ((ix < nx) && (iy < ny)){

      const float pi = 4*atan(1);
      float pix_y = y0 + dy*(iy+0.5);
      float pix_x = x0 + dx*(ix+0.5);

      float frad = sqrt(pix_x*pix_x + pix_y*pix_y);

      if (frad<=fov_radius){

        float fphi = atan2(pix_y,pix_x);

        for(int sindex = 0; sindex < nviews; sindex++){

            float s = sindex*ds;

            // location of the source
            float xsource = radius*cos(s);
            float ysource = radius*sin(s);

            // detector center
            float xDetCenter = (radius - source_to_detector)*cos(s);
            float yDetCenter = (radius - source_to_detector)*sin(s);

            // unit vector in the direction of the detector line
            float eux = -sin(s);
            float euy =  cos(s);

            //Unit vector in the direction perpendicular to the detector line
            float ewx = cos(s);
            float ewy = sin(s);

            float bigu = (radius+frad*sin(s-fphi-pi/2.0))/radius;
            float bpweight = 1.0/(bigu*bigu);

            float ew_dot_source_pix = (pix_x-xsource)*ewx + (pix_y-ysource)*ewy;
            float rayratio = -source_to_detector/ew_dot_source_pix;

            float det_int_x = xsource+rayratio*(pix_x-xsource);
            float det_int_y = ysource+rayratio*(pix_y-ysource);

            float upos = ((det_int_x-xDetCenter)*eux +(det_int_y-yDetCenter)*euy);

            if ((upos-u0 >= du/2.0) && (upos-u0 < detectorlength-du/2.0)){
              float bin_loc = (upos-u0)/du + 0.5;
              int nbin1 = static_cast<int>(bin_loc)-1;
              int nbin2 = nbin1+1;
              float frac = bin_loc - static_cast<int>(bin_loc);
              auto pix_value = image[ib][ix][iy];

              atomicAdd(&sinogram[ib][sindex][nbin1],(1.0f-frac)*bpweight*ds*pix_value);
              atomicAdd(&sinogram[ib][sindex][nbin2],frac*bpweight*ds*pix_value);

              // float det_value = frac*sinogram[ib][sindex][nbin2]+(1.0-frac)*sinogram[ib][sindex][nbin1];
              // atomicAdd(&image[ib][ix][iy],bpweight*det_value*ds);
            }
         }
      }
   }
}

// computes pixel-driven backprojetion over one view
__global__ void backprojection_wpd_kernel(torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> image,
                                          const torch::PackedTensorAccessor32<float,3,torch::RestrictPtrTraits> sinogram,
                                          const float dx,
                                          const float dy,
                                          const float x0,
                                          const float y0,
                                          const float fanangle2,
                                          const float detectorlength,
                                          const float u0,
                                          const float du,
                                          const float ds,
                                          const float radius,
                                          const float source_to_detector,
                                          const float fov_radius,
                                          const int nbins,
                                          const int nviews,
                                          const int nx,
                                          const int ny){

    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int ib = blockIdx.z; //batch index

    if ((ix < nx) && (iy < ny)){

      const float pi = 4*atan(1);
      const float pix_y = y0 + dy*(iy+0.5);
      const float pix_x = x0 + dx*(ix+0.5);

      const float frad = sqrt(pix_x*pix_x + pix_y*pix_y);

      if (frad<=fov_radius){

        const float fphi = atan2(pix_y,pix_x);

        for(int sindex = 0; sindex < nviews; sindex++){

            float s = sindex*ds;

            // location of the source
            float xsource = radius*cos(s);
            float ysource = radius*sin(s);

            // detector center
            float xDetCenter = (radius - source_to_detector)*cos(s);
            float yDetCenter = (radius - source_to_detector)*sin(s);

            // unit vector in the direction of the detector line
            float eux = -sin(s);
            float euy =  cos(s);

            //Unit vector in the direction perpendicular to the detector line
            float ewx = cos(s);
            float ewy = sin(s);

            float bigu = (radius+frad*sin(s-fphi-pi/2.0))/radius;
            float bpweight = 1.0/(bigu*bigu);

            float ew_dot_source_pix = (pix_x-xsource)*ewx + (pix_y-ysource)*ewy;
            float rayratio = -source_to_detector/ew_dot_source_pix;

            float det_int_x = xsource+rayratio*(pix_x-xsource);
            float det_int_y = ysource+rayratio*(pix_y-ysource);

            float upos = ((det_int_x-xDetCenter)*eux +(det_int_y-yDetCenter)*euy);

            if ((upos-u0 >= du/2.0) && (upos-u0 < detectorlength-du/2.0)){
              float bin_loc = (upos-u0)/du + 0.5;
              int nbin1 = static_cast<int>(bin_loc)-1;
              int nbin2 = nbin1+1;
              float frac= bin_loc - static_cast<int>(bin_loc);
              float det_value = frac*sinogram[ib][sindex][nbin2]+(1.0-frac)*sinogram[ib][sindex][nbin1];
              atomicAdd(&image[ib][ix][iy],bpweight*det_value*ds);
            }
         }
      }
   }
}

torch::Tensor circularFanbeamProjection_cuda(const torch::Tensor image, const int nx, const int ny, const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
    const float dx = ximageside/nx;
    const float dy = yimageside/ny;
    const float x0 = -ximageside/2.0;
    const float y0 = -yimageside/2.0;

    // compute length of detector so that it views the inscribed FOV of the image array
    const float fanangle2 = std::asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
    const float detectorlength = 2.0*std::tan(fanangle2)*source_to_detector;
    const float u0 = -detectorlength/2.0;

    const float du = detectorlength/nbins;
    const float ds = slen/nviews;

    const auto image_a = image.packed_accessor32<float,3,torch::RestrictPtrTraits>();
    const int batch_size = image_a.size(0); //batch_size

    const auto options = torch::TensorOptions().dtype(image.dtype()).device(image.device());
    auto sinogram = torch::zeros({batch_size, nviews, nbins}, options);
    auto sinogram_a = sinogram.packed_accessor32<float,3,torch::RestrictPtrTraits>();

    // parallize over rays, batches
    dim3 block_dim(16, 16);
    dim3 grid_dim(roundup_div(nbins, 16), roundup_div(nviews, 16), batch_size );

    projection_view_kernel<<<grid_dim, block_dim>>>(image_a,
                                                sinogram_a,
                                                dx,
                                                dy,
                                                x0,
                                                y0,
                                                fanangle2,
                                                detectorlength,
                                                u0,
                                                du,
                                                ds,
                                                radius,
                                                source_to_detector,
                                                nbins,
                                                nviews,
                                                nx,
                                                ny);

    return sinogram;
}

// exact matrix transpose of circularFanbeamProjection
torch::Tensor circularFanbeamBackProjection_cuda(const torch::Tensor sinogram, const int nx, const int ny,
                              const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
   const float dx = ximageside/nx;
   const float dy = yimageside/ny;
   const float x0 = -ximageside/2.0;
   const float y0 = -yimageside/2.0;

   // compute length of detector so that it views the inscribed FOV of the image array
   const float fanangle2 = std::asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
   const float detectorlength = 2.0*std::tan(fanangle2)*source_to_detector;
   const float u0 = -detectorlength/2.0;

   const float du = detectorlength/nbins;
   const float ds = slen/nviews;

   const float fov_radius = ximageside/2.0;

   const auto sinogram_a = sinogram.packed_accessor32<float,3,torch::RestrictPtrTraits>();
   const int batch_size = sinogram_a.size(0); //batch_size

   const auto options = torch::TensorOptions().dtype(sinogram.dtype()).device(sinogram.device());
   auto image = torch::zeros({batch_size, nx, ny}, options);
   auto image_a = image.packed_accessor32<float,3,torch::RestrictPtrTraits>();

   // const int threads = nviews; //one per view, max 1024 -- todo: add input validation
   // const int blocks = batch_size; //match to batch size

   // parallize over rays, batches
   dim3 block_dim(16, 16);
   dim3 grid_dim(roundup_div(nbins, 16), roundup_div(nviews, 16), batch_size );

   backprojection_view_kernel<<<grid_dim, block_dim>>>(image_a,
                                               sinogram_a,
                                               dx,
                                               dy,
                                               x0,
                                               y0,
                                               fanangle2,
                                               detectorlength,
                                               u0,
                                               du,
                                               ds,
                                               radius,
                                               source_to_detector,
                                               fov_radius,
                                               nbins,
                                               nviews,
                                               nx,
                                               ny);

    return image;
}

torch::Tensor circularFanbeamWPDProjection_cuda(const torch::Tensor image, const int nx, const int ny,
                              const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
   const float dx = ximageside/nx;
   const float dy = yimageside/ny;
   const float x0 = -ximageside/2.0;
   const float y0 = -yimageside/2.0;

   // compute length of detector so that it views the inscribed FOV of the image array
   const float fanangle2 = asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
   const float detectorlength = 2.0*tan(fanangle2)*source_to_detector;
   const float u0 = -detectorlength/2.0;

   const float du = detectorlength/nbins;
   const float ds = slen/nviews;

   const float fov_radius = ximageside/2.0;

   const auto image_a = image.packed_accessor32<float,3,torch::RestrictPtrTraits>();
   const int batch_size = image_a.size(0); //batch_size

   const auto options = torch::TensorOptions().dtype(image.dtype()).device(image.device());
   auto sinogram = torch::zeros({batch_size, nviews, nbins}, options);
   auto sinogram_a = sinogram.packed_accessor32<float,3,torch::RestrictPtrTraits>();

   // parallize over pixels
   dim3 block_dim(16, 16);
   dim3 grid_dim(roundup_div(nx, 16), roundup_div(ny, 16), batch_size);

   projection_wpd_kernel<<<grid_dim, block_dim>>>(image_a,
                                                   sinogram_a,
                                                   dx,
                                                   dy,
                                                   x0,
                                                   y0,
                                                   fanangle2,
                                                   detectorlength,
                                                   u0,
                                                   du,
                                                   ds,
                                                   radius,
                                                   source_to_detector,
                                                   fov_radius,
                                                   nbins,
                                                   nviews,
                                                   nx,
                                                   ny);
   return sinogram;
}

torch::Tensor circularFanbeamWPDBackProjection_cuda(const torch::Tensor sinogram, const int nx, const int ny,
                              const float ximageside, const float yimageside,
                              const float radius, const float source_to_detector,
                              const int nviews, const float slen, const int nbins) {
   const float dx = ximageside/nx;
   const float dy = yimageside/ny;
   const float x0 = -ximageside/2.0;
   const float y0 = -yimageside/2.0;

   // compute length of detector so that it views the inscribed FOV of the image array
   const float fanangle2 = asin((ximageside/2.0)/radius);  //This only works for ximageside = yimageside
   const float detectorlength = 2.0*tan(fanangle2)*source_to_detector;
   const float u0 = -detectorlength/2.0;

   const float du = detectorlength/nbins;
   const float ds = slen/nviews;

   const float fov_radius = ximageside/2.0;

   const auto sinogram_a = sinogram.packed_accessor32<float,3,torch::RestrictPtrTraits>();
   const int batch_size = sinogram_a.size(0); //batch_size

   const auto options = torch::TensorOptions().dtype(sinogram.dtype()).device(sinogram.device());
   auto image = torch::zeros({batch_size, nx, ny}, options);
   auto image_a = image.packed_accessor32<float,3,torch::RestrictPtrTraits>();

   // parallize over pixels
   dim3 block_dim(16, 16);
   dim3 grid_dim(roundup_div(nx, 16), roundup_div(ny, 16), batch_size);

   backprojection_wpd_kernel<<<grid_dim, block_dim>>>(image_a,
                                                   sinogram_a,
                                                   dx,
                                                   dy,
                                                   x0,
                                                   y0,
                                                   fanangle2,
                                                   detectorlength,
                                                   u0,
                                                   du,
                                                   ds,
                                                   radius,
                                                   source_to_detector,
                                                   fov_radius,
                                                   nbins,
                                                   nviews,
                                                   nx,
                                                   ny);
   return image;
}
